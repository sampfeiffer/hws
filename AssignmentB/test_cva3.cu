#include "hip/hip_runtime.h"
// calculate how many deals i can read in to one gpu.
// create a device_vector of the appropriate amount of counterparties
// run the simulations and cva calculator on the vector of counterparties

#include <stdio.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <omp.h>
#include <stdio.h>
#include <stdlib.h>
#include <thrust/host_vector.h>
#include <thrust/device_vector.h>
#include <thrust/sort.h>
#include <thrust/copy.h>
#include <time.h>
#include <sys/time.h>

#include <vector>
#include "parameters.h"
#include "counterparty.h"
#include "state.h"

//#define DSIZE 1000

using namespace std;

struct calculate_cva{
    Parameters params;
    int num_of_steps;

    calculate_cva(Parameters params_, int num_of_steps_) : params(params_), num_of_steps(num_of_steps_)
    {}
    __device__ __host__
    float operator()(Counterparty &cp) {
        float cva=0;
        float total_value;
        State world_state(params);
        for (int i=0; i<num_of_steps; ++i){
            total_value = 0;
            world_state.sim_next_step();
            // CVA for fx
            for (unsigned int fx=0; fx<cp.num_of_fx; ++fx){
                total_value += max(cp.fx_deals[fx]->value(world_state.fx_rate),float(0.0));
            }
            // CVA for swaps
            for (unsigned int sw=0; sw<cp.num_of_swap; ++sw){
                total_value += max(cp.swap_deals[sw]->value(world_state),float(0.0));
            }
            cva += world_state.cva_disc_factor * cp.prob_default(world_state.time) * total_value;
        }
        cva *= 1-params.recovery_rate;
        return cva;
    }
};

int main(int argc, char *argv[])
{
    clock_t program_start_time, end_time;
    program_start_time = clock();

    const char* parameters_filename="parameters.txt";
    const char* state0_filename="state0.txt";
    const char* hazard_buckets_filename="hazard_buckets.dat";
    const char* counterparty_deals_filename="counterparty_deals.dat";
    const char* fx_details_filename="fx_details.dat";
    const char* swap_details_filename="swap_details.dat";
    std::ifstream counterparty_deals_infile, fx_details_infile, swap_details_infile, hazard_buckets_infile;

    // Get parameters and initial state of the world.
    Parameters params(parameters_filename, state0_filename);
    //params.print();

    // Get the list of hazard rate bucket endpoints
    int hazard_buckets[5];
    hazard_buckets_infile.open(hazard_buckets_filename);
    if (!hazard_buckets_infile.is_open()){
        std::cout << "ERROR: hazard_buckets.dat file could not be opened. Exiting.\n";
        exit(1);
    }
    for (int i=0; i<5; ++i) hazard_buckets_infile >> hazard_buckets[i];
    hazard_buckets_infile.close();


    // Open the counterparty deals and deal details
    counterparty_deals_infile.open(counterparty_deals_filename);
    if (!counterparty_deals_infile.is_open()){
        std::cout << "ERROR: counterparty_deals.dat file could not be opened. Exiting.\n";
        exit(1);
    }
    fx_details_infile.open(fx_details_filename);
    if (!fx_details_infile.is_open()){
        std::cout << "ERROR: fx_details.dat file could not be opened. Exiting.\n";
        exit(1);
    }
    swap_details_infile.open(swap_details_filename);
    if (!swap_details_infile.is_open()){
        std::cout << "ERROR: swap_details.dat file could not be opened. Exiting.\n";
        exit(1);
    }

    //thrust::device_vector<Counterparty> cp_vector;
    thrust::host_vector<Counterparty> cp_vector;
    //std::vector<Counterparty> cp_vector;

    // Read deals into memory
    int current_id=1, deal_id, id=1, deals_handled=0, bucket=0;
    float hazard_rate=0.10;

    int fx_id, swap_id, notional, tenor, start_of_data, fx_count, swap_count;
    char position, denomination;
    float fixed_rate;
    counterparty_deals_infile >> deal_id;

    while (deals_handled <= params.deals_at_once){
        if (id > hazard_buckets[bucket]){
            ++bucket;
            hazard_rate -= 0.02;
        }
        start_of_data = counterparty_deals_infile.tellg();
        fx_count = 0;
        swap_count = 0;

        do{
            counterparty_deals_infile >> deal_id;
            if (deal_id<params.fx_num) ++fx_count;
            else ++swap_count;
            counterparty_deals_infile >> current_id;
        } while(current_id == id);
        counterparty_deals_infile.seekg(start_of_data,counterparty_deals_infile.beg);

        Counterparty cp(id, hazard_rate, fx_count, swap_count);
        do{
            counterparty_deals_infile >> deal_id;
            if (deal_id<params.fx_num){
                fx_details_infile >> fx_id;
                fx_details_infile >> notional;
                fx_details_infile >> position;
                cp.add_fx(fx_id, notional, position);
            }
            else {
                swap_details_infile >> swap_id;
                swap_details_infile >> denomination;
                swap_details_infile >> notional;
                swap_details_infile >> fixed_rate;
                swap_details_infile >> tenor;
                swap_details_infile >> position;
                cp.add_swap(swap_id, denomination, notional, fixed_rate, tenor, position);
            }
            ++deals_handled;
            counterparty_deals_infile >> current_id;
        } while(current_id == id);
        cp_vector.push_back(cp);
        ++id;
    }

    int num_of_steps = params.days_in_year*params.time_horizon/params.step_size;

    // determine the number of CUDA capable GPUs
    int num_gpus = 0;
    hipGetDeviceCount(&num_gpus);
    if (num_gpus < 1)
    {
        printf("no CUDA capable devices were detected\n");
        return 1;
    }
    printf("number of host CPUs:\t%d\n", omp_get_num_procs());
    printf("number of CUDA devices:\t%d\n", num_gpus);
    for (int i = 0; i < num_gpus; i++)
    {
        hipDeviceProp_t dprop;
        hipGetDeviceProperties(&dprop, i);
        printf("   %d: %s\n", i, dprop.name);
    }
    int simulations_per_gpu = params.simulation_num/num_gpus;

    // initialize data
    typedef thrust::device_vector<Counterparty> dvec;
    typedef thrust::device_vector<float> cva_vector;
    typedef dvec *p_dvec;
    typedef cva_vector *p_cva_vec;
    std::vector<p_dvec> dvecs;
    std::vector<p_cva_vec> cva_vectors_std;

    for(unsigned int i = 0; i < num_gpus; i++) {
        hipSetDevice(i);
        p_dvec temp = new dvec(cp_vector.size());
        dvecs.push_back(temp);
        p_cva_vec temp2 = new cva_vector(cp_vector.size());
        cva_vectors_std.push_back(temp2);
    }

    //thrust::host_vector<int> data(DSIZE);
    //thrust::generate(data.begin(), data.end(), rand);

    // copy data
    for (unsigned int i = 0; i < num_gpus; i++) {
        hipSetDevice(i);
        thrust::copy(cp_vector.begin(), cp_vector.end(), (*(dvecs[i])).begin());
    }

    // run as many CPU threads as there are CUDA devices
    omp_set_num_threads(num_gpus);  // create as many CPU threads as there are CUDA devices
    #pragma omp parallel
    {
        unsigned int cpu_thread_id = omp_get_thread_num();
        hipSetDevice(cpu_thread_id);
        thrust::transform((*(dvecs[cpu_thread_id])).begin(), (*(dvecs[cpu_thread_id])).end(), (*(cva_vectors_std[cpu_thread_id])).begin(), calculate_cva(params, num_of_steps));
        hipDeviceSynchronize();
    }


    counterparty_deals_infile.close();
    fx_details_infile.close();
    swap_details_infile.close();

    end_time = clock() - program_start_time;
    std::cout << "Timing: whole program " << float(end_time)/CLOCKS_PER_SEC << " seconds.\n";

    std::cout << "\n";

    return 0;
}
