#include <stdio.h>
#include <hip/hip_runtime.h>

int main(void) {

    hipDeviceProp_t deviceProp1;
    hipDeviceProp_t deviceProp2;

    hipGetDeviceProperties(&deviceProp1, 0);
    hipGetDeviceProperties(&deviceProp2, 1);

    printf("\nDevice 0 has %f MB of global RAM, while Device 1 has %f. Cheers!\n",
            deviceProp1.totalGlobalMem / (1024. * 1024.),
            deviceProp2.totalGlobalMem / (1024. * 1024.));

    int num_gpus=0;
    hipGetDeviceCount(&num_gpus);
    printf("number of CUDA devices:\t%d\n", num_gpus);

    return 0;
}
