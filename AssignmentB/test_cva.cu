#include <stdio.h>
#include <hip/hip_runtime.h>

int main(void) {

    hipDeviceProp_t deviceProp1;
    hipDeviceProp_t deviceProp2;

    hipGetDeviceProperties(&deviceProp1, 0);
    hipGetDeviceProperties(&deviceProp2, 1);

    printf("\nDevice 0 has %f MB of global RAM, while Device 1 has %f. Cheers!",
            deviceProp1.totalGlobalMem / (1024. * 1024.),
            deviceProp2.totalGlobalMem / (1024. * 1024.));

    return 0;
}
