// calculate how many deals i can read in to one gpu.
// create a device_vector of the appropriate amount of counterparties
// run the simulations and cva calculator on the vector of counterparties


struct add
{
    T operator()(T a, T b)
    {
        return a + b;
    }
};

#include <stdio.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

int main(void) {

    hipDeviceProp_t deviceProp1;
    hipDeviceProp_t deviceProp2;

    hipGetDeviceProperties(&deviceProp1, 0);
    hipGetDeviceProperties(&deviceProp2, 1);

    printf("\nDevice 0 has %f MB of global RAM, while Device 1 has %f. Cheers!\n",
            deviceProp1.totalGlobalMem / (1024. * 1024.),
            deviceProp2.totalGlobalMem / (1024. * 1024.));

    int num_gpus=0;
    hipGetDeviceCount(&num_gpus);
    printf("number of CUDA devices:\t%d\n", num_gpus);

    int x = 10; int y = 20; int z;

    add<int> func; // create an add functor for T=int

    z = func(x,y); // invoke functor on x and y
    float x = 10; float y = 20; float z;
    add<float> func; // create an add functor for T=float
    z = func(x,y); // invoke functor on x and y



    return 0;
}
