#include "hip/hip_runtime.h"
#include <thrust/host_vector.h>
#include <thrust/device_vector.h>

#include <vector>
#include <time.h>
#include "parameters.h"
#include "counterparty.h"
#include "state.h"
#include "data_reader.h"

struct calculate_cva{
    Parameters params;
    int num_of_steps;

    calculate_cva(Parameters params_, int num_of_steps_) : params(params_), num_of_steps(num_of_steps_)
    {}
    __device__ __host__
    float operator()(Counterparty &cp) {
        float cva=0;
        float total_value;
        State world_state(params);
        for (int i=0; i<num_of_steps; ++i){
            total_value = 0;
            world_state.sim_next_step();
            // CVA for fx
            for (unsigned int fx=0; fx<cp.num_of_fx; ++fx){
                total_value += max(cp.fx_deals[fx]->value(world_state.fx_rate),float(0.0));
            }
            // CVA for swaps
            for (unsigned int sw=0; sw<cp.num_of_swap; ++sw){
//                total_value += max(cp.swap_deals[sw]->value(world_state),float(0.0));
            }
            cva += world_state.cva_disc_factor * cp.prob_default(world_state.time) * total_value;
        }
        cva *= 1-params.recovery_rate;
        return cva;
    }
};

int main(int argc, char *argv[])
{
    clock_t program_start_time, end_time;
    program_start_time = clock();

    const char* parameters_filename="parameters.txt";
    const char* state0_filename="state0.txt";

    // Get parameters and initial state of the world.
    Parameters params(parameters_filename, state0_filename);
    //params.print();

    // Get counterparty data and store info in cp_vector_temp
    thrust::device_vector<Counterparty> cp_vector_temp;
    Data_reader data;
    data.get_next_data(cp_vector_temp, params);
    //std::cout << "test info " << cp_vector_temp[1].fx_deals[0]->fx_id << "\n";

    thrust::device_vector<Counterparty> cp_vector(cp_vector_temp.begin(), cp_vector_temp.end());

    int num_of_steps = params.days_in_year*params.time_horizon/params.step_size;

    end_time = clock() - program_start_time;
    std::cout << "Timing: whole program " << float(end_time)/CLOCKS_PER_SEC << " seconds.\n";

    //thrust::host_vector<float> cva_vector(cp_vector.size());
    thrust::device_vector<float> cva_vector(cp_vector.size());
    std::cout << "here1 " << cva_vector.size() << "\n";
    thrust::transform(cp_vector.begin(), cp_vector.end(), cva_vector.begin(), calculate_cva(params, num_of_steps));
    std::cout << "here2\n";
    std::cout << "size1 " << cva_vector.size() << "\n";
    thrust::host_vector<float> cva_vector_host(cva_vector);
    std::cout << "size2 " << cva_vector_host.size() << "\n";

    for (unsigned int i=0; i<cva_vector_host.size(); ++i){
        std::cout << "here in " << i+1 << "\n";
        std::cout << "cva " << i+1 << " " << cva_vector_host[i] << "\n";
    }

    data.close_files();
    std::cout << "\n";
    return 0;
}

