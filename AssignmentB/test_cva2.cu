#include "hip/hip_runtime.h"
#include <thrust/host_vector.h>
#include <thrust/device_vector.h>
#include <thrust/random/linear_congruential_engine.h>
#include <thrust/random/normal_distribution.h>

#include <vector>
#include <algorithm>
#include "parameters.h"
#include "counterparty.h"
#include "state.h"

struct calculate_cva{
    Parameters params;

    calculate_cva(Parameters &params_) : params(params_)
    {}
    __host__
    float operator()(Counterparty &cp) {
        float temp1;
        float temp2;
        float cva=0;
        int num_of_steps = 360*params.time_horizon/params.step_size;
        State world_state(params);
        for (int i=0; i<num_of_steps; ++i){
            world_state.sim_next_step();
            // CVA for fx
            for (unsigned int fx=0; fx<cp.num_of_fx; ++fx){
                temp1 = world_state.cva_disc_factor;
                temp2 = cp.prob_default(world_state.time);
                cva += temp1*temp2* std::max(cp.fx_deals[fx]->value(world_state.fx_rate_beg, world_state.fx_rate),0.0);
            }
            // CVA for swaps
            for (unsigned int sw=0; sw<cp.num_of_swap; ++sw){
                cva += world_state.cva_disc_factor * cp.prob_default(world_state.time)
                       * std::max(cp.swap_deals[sw]->value(world_state),0.0);
            }
        }
        cva *= 1-params.recovery_rate;
        return cva;
    }
};

int main(int argc, char *argv[])
{
    const char* parameters_filename="parameters.txt";
    const char* state0_filename="state0.txt";
    const char* hazard_buckets_filename="hazard_buckets.txt";
    const char* counterparty_deals_filename="counterparty_deals.txt";
    const char* fx_details_filename="fx_details.txt";
    const char* swap_details_filename="swap_details.txt";
    std::ifstream counterparty_deals_infile, fx_details_infile, swap_details_infile, hazard_buckets_infile;

    // Get parameters and initial state of the world.
    Parameters params(parameters_filename, state0_filename);
    params.print();

    // Get the list of hazard rate bucket endpoints
    int hazard_buckets[5];
    hazard_buckets_infile.open(hazard_buckets_filename);
    if (!hazard_buckets_infile.is_open()){
        std::cout << "ERROR: hazard_buckets.txt file could not be opened. Exiting.\n";
        exit(1);
    }
    for (int i=0; i<5; ++i) hazard_buckets_infile >> hazard_buckets[i];
    hazard_buckets_infile.close();


    // Open the counterparty deals and deal details
    counterparty_deals_infile.open(counterparty_deals_filename);
    if (!counterparty_deals_infile.is_open()){
        std::cout << "ERROR: counterparty_deals.txt file could not be opened. Exiting.\n";
        exit(1);
    }
    fx_details_infile.open(fx_details_filename);
    if (!fx_details_infile.is_open()){
        std::cout << "ERROR: fx_details.txt file could not be opened. Exiting.\n";
        exit(1);
    }
    swap_details_infile.open(swap_details_filename);
    if (!swap_details_infile.is_open()){
        std::cout << "ERROR: swap_details.txt file could not be opened. Exiting.\n";
        exit(1);
    }

    // Read deals into memory
    int current_id=1, deal_id, id=1, deals_handled=0, bucket=0;
    float hazard_rate=0.10;

    //thrust::device_vector<Counterparty> cp_vector;
    //thrust::device_vector<float> cva_vector;
    thrust::host_vector<Counterparty> cp_vector;
    thrust::host_vector<float> cva_vector;
    //std::vector<Counterparty> cp_vector;
    //std::vector<float> cva_vector;

    int fx_id, swap_id, notional, tenor, start_of_data, fx_count, swap_count;
    char position, denomination;
    float fixed_rate;
    counterparty_deals_infile >> deal_id;

    while (deals_handled <= params.deals_at_once){
        if (id > hazard_buckets[bucket]){
            ++bucket;
            hazard_rate -= 0.02;
        }
        start_of_data = counterparty_deals_infile.tellg();
        fx_count = 0;
        swap_count = 0;

        do{
            counterparty_deals_infile >> deal_id;
            if (deal_id<params.fx_num) ++fx_count;
            else ++swap_count;
            counterparty_deals_infile >> current_id;
        } while(current_id == id);
        counterparty_deals_infile.seekg(start_of_data,counterparty_deals_infile.beg);
        //std::cout << id << " " << fx_count << " " << swap_count << "\n";

        Counterparty cp(id, hazard_rate, fx_count, swap_count);
        do{
            counterparty_deals_infile >> deal_id;
            if (deal_id<params.fx_num){
                fx_details_infile >> fx_id;
                fx_details_infile >> notional;
                fx_details_infile >> position;
                cp.add_fx(fx_id, notional, position);
            }
            else {
                swap_details_infile >> swap_id;
                swap_details_infile >> denomination;
                swap_details_infile >> notional;
                swap_details_infile >> fixed_rate;
                swap_details_infile >> tenor;
                swap_details_infile >> position;
                cp.add_swap(swap_id, denomination, notional, fixed_rate, tenor, position);
            }
            ++deals_handled;
            counterparty_deals_infile >> current_id;
        } while(current_id == id);
        cp_vector.push_back(cp);
        ++id;
    }

    //int num_of_steps = 360*params.time_horizon/params.step_size;

    thrust::transform(cp_vector.begin(), cp_vector.end(), cva_vector.begin(), calculate_cva(params));
    //std::transform(cp_vector.begin(), cp_vector.end(), cva_vector.begin(), calculate_cva(params));


    //for (unsigned int i=0; i<cp_vector.size(); ++i){
    //    std::cout << "cva " << i+1 << " " << cp_vector[i].cva << "\n";
    //}

    for (unsigned int i=0; i<cva_vector.size(); ++i){
        std::cout << "cva " << i+1 << " " << cva_vector[i] << "\n";
    }


    counterparty_deals_infile.close();
    fx_details_infile.close();
    swap_details_infile.close();

    std::cout << "\n";


    return 0;
}

