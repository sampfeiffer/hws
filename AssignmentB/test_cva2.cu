#include "hip/hip_runtime.h"
#include <thrust/host_vector.h>
#include <thrust/device_vector.h>
#include <thrust/random/linear_congruential_engine.h>
#include <thrust/random/normal_distribution.h>

#include <vector>
#include "parameters.h"
#include "counterparty.h"
#include "state.h"

int main(int argc, char *argv[])
{
    const char* parameters_filename="parameters.txt";
    const char* state0_filename="state0.txt";
    const char* hazard_buckets_filename="hazard_buckets.txt";
    const char* counterparty_deals_filename="counterparty_deals.txt";
    const char* fx_details_filename="fx_details.txt";
    const char* swap_details_filename="swap_details.txt";
    std::ifstream counterparty_deals_infile, fx_details_infile, swap_details_infile, hazard_buckets_infile;

    // Get parameters and initial state of the world.
    Parameters params(parameters_filename, state0_filename);
    params.print();

    // Get the list of hazard rate bucket endpoints
    int hazard_buckets[5];
    hazard_buckets_infile.open(hazard_buckets_filename);
    if (!hazard_buckets_infile.is_open()){
        std::cout << "ERROR: hazard_buckets.txt file could not be opened. Exiting.\n";
        exit(1);
    }
    for (int i=0; i<5; ++i) hazard_buckets_infile >> hazard_buckets[i];
    hazard_buckets_infile.close();


    // Open the counterparty deals and deal details
    counterparty_deals_infile.open(counterparty_deals_filename);
    if (!counterparty_deals_infile.is_open()){
        std::cout << "ERROR: counterparty_deals.txt file could not be opened. Exiting.\n";
        exit(1);
    }
    fx_details_infile.open(fx_details_filename);
    if (!fx_details_infile.is_open()){
        std::cout << "ERROR: fx_details.txt file could not be opened. Exiting.\n";
        exit(1);
    }
    swap_details_infile.open(swap_details_filename);
    if (!swap_details_infile.is_open()){
        std::cout << "ERROR: swap_details.txt file could not be opened. Exiting.\n";
        exit(1);
    }

    // Read deals into memory
    int current_id=1, deal_id, id=1, deals_handled=0, bucket=0;
    float hazard_rate=0.10;

    //thrust::device_vector<Counterparty> cp_vector;
    std::vector<Counterparty> cp_vector;

    int fx_id, swap_id, notional, tenor, start_of_data, fx_count, swap_count;
    char position, denomination;
    float fixed_rate;
    counterparty_deals_infile >> deal_id;

    while (deals_handled <= params.deals_at_once){
        if (id > hazard_buckets[bucket]){
            ++bucket;
            hazard_rate -= 0.02;
        }
        start_of_data = counterparty_deals_infile.tellg();
        fx_count = 0;
        swap_count = 0;

        do{
            counterparty_deals_infile >> deal_id;
            if (deal_id<params.fx_num) ++fx_count;
            else ++swap_count;
            counterparty_deals_infile >> current_id;
        } while(current_id == id);
        counterparty_deals_infile.seekg(start_of_data,counterparty_deals_infile.beg);
        //std::cout << id << " " << fx_count << " " << swap_count << "\n";

        Counterparty cp(id, hazard_rate, fx_count, swap_count);
        do{
            counterparty_deals_infile >> deal_id;
            if (deal_id<params.fx_num){
                fx_details_infile >> fx_id;
                fx_details_infile >> notional;
                fx_details_infile >> position;
                cp.add_fx(fx_id, notional, position);
            }
            else {
                swap_details_infile >> swap_id;
                swap_details_infile >> denomination;
                swap_details_infile >> notional;
                swap_details_infile >> fixed_rate;
                swap_details_infile >> tenor;
                swap_details_infile >> position;
                cp.add_swap(swap_id, denomination, notional, fixed_rate, tenor, position);
            }
            ++deals_handled;
            counterparty_deals_infile >> current_id;
        } while(current_id == id);
        cp_vector.push_back(cp);
        ++id;
    }

//    std::cout << "size " << cp_vector.size() << "\n";
//    for (unsigned int i=0; i<cp_vector.size(); ++i){
//        cp_vector[i].print();
//    }

    int num_of_steps = 360*params.time_horizon/params.step_size;

    // Generate a state of the world that will be changed through time
    State world_state(params);
    std::vector<State> state_vector;
    state_vector.push_back(world_state);
    for (int i=0; i<num_of_steps; ++i){
        world_state.sim_next_step();
        State next_state = world_state;
        state_vector.push_back(next_state);
    }

    // Calculate CVA
    for (unsigned int cp=0; cp<cp_vector.size(); ++cp){
        // CVA for fx
        for (unsigned int fx=0; fx<cp_vector[cp].num_of_fx; ++fx){
            for (unsigned int i=0; i<state_vector.size(); ++i){
                cp_vector[cp].cva += state_vector[i].cva_disc_factor * cp_vector[cp].prob_default(state_vector[i].time)
                                     * std::max(cp_vector[cp].fx_deals[fx]->value(state_vector[i].fx_rate_beg, state_vector[i].fx_rate),0.0);
            }
        }
        // CVA for swaps
        for (unsigned int sw=0; sw<cp_vector[cp].num_of_swap; ++sw){
            for (unsigned int i=0; i<state_vector.size(); ++i){
                cp_vector[cp].cva += state_vector[i].cva_disc_factor * cp_vector[cp].prob_default(state_vector[i].time)
                                     * std::max(cp_vector[cp].swap_deals[sw]->value(state_vector[i]),0.0);
            }
        }
        cp_vector[cp].cva *= 1-params.recovery_rate;
    }

    for (unsigned int i=0; i<cp_vector.size(); ++i){
        std::cout << "cva " << i+1 << " " << cp_vector[i].cva << "\n";
    }


    counterparty_deals_infile.close();
    fx_details_infile.close();
    swap_details_infile.close();

    std::cout << "\n";


    return 0;
}
