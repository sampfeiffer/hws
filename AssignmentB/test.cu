#include "hip/hip_runtime.h"
#include <thrust/random.h>
#include <thrust/count.h>

#include <iostream>

const int N = 1000000; // Number of Monte-Carlo simulations.

using namespace std;


struct random_point {
private:
    thrust::minstd_rand rng;
public:
    __device__ // __host__
    float2 operator()(int index) {
        rng.discard(2*index);
        return make_float2(
            (float)rng() / thrust::minstd_rand::max,
            (float)rng() / thrust::minstd_rand::max);
    }
};

struct inside_circle {
private:
    __device__ // __host__
    unsigned int inside(float2 p) const {
        return (((p.x-0.5)*(p.x-0.5)+(p.y-0.5)*(p.y-0.5))<0.25) ? 1 : 0;
    }
public:
    // Used for-on-the fly.
    __device__ // __host__
    unsigned int operator()(int index) const {
        // Generate a random point.
        random_point point;
        return inside(point(index));
    }
};

int main()
{
// DEVICE: Calculations done on GPU.
thrust::counting_iterator<int> index(0);
size_t total = thrust::count_if(index, index+N, inside_circle());

// HOST: Print estimate of PI.
cout << "PI: " << 4.0*(float)total/(float)N << endl;

return 0;
}
